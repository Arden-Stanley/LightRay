#include "Renderer.h"

#include "RaytracingKernel.cuh"
#include <glad/glad.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>


namespace LR {
    static hipGraphicsResource *m_texPtr;
    static hipArray_t m_mappedTex;
    static hipSurfaceObject_t surf;

    Renderer::Renderer(unsigned int texHandle, int screenWidth, int screenHeight) 
    : m_texHandle(texHandle), m_screenWidth(screenWidth), m_screenHeight(screenHeight) {

        hipGraphicsGLRegisterImage(&m_texPtr, m_texHandle, GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore);
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            std::cerr << "CUDA Kernel Error 1: " << hipGetErrorString(err) << "\n";
        }

    }

    Renderer::~Renderer() {
        hipGraphicsUnregisterResource(m_texPtr);
    }

    void Renderer::render() {
        hipGraphicsMapResources(1, &m_texPtr, 0);
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            std::cerr << "CUDA Kernel Error 2: " << hipGetErrorString(err) << "\n";
        }
        hipGraphicsSubResourceGetMappedArray(&m_mappedTex, m_texPtr, 0, 0);
        err = hipGetLastError();
        if (err != hipSuccess) {
            std::cerr << "CUDA Kernel Error 3: " << hipGetErrorString(err) << "\n";
        }



        hipResourceDesc resDesc = {};
        //memset(&resDesc, 0, sizeof(resDesc));
        resDesc.resType = hipResourceTypeArray;
        resDesc.res.array.array = m_mappedTex;
        err = hipGetLastError();
        if (err != hipSuccess) {
            std::cerr << "CUDA Kernel Error 4: " << hipGetErrorString(err) << "\n";
        }

        
        hipCreateSurfaceObject(&surf, &resDesc);
        err = hipGetLastError();
        if (err != hipSuccess) {
            std::cerr << "CUDA Kernel Error 5: " << hipGetErrorString(err) << "\n";
        }

        
        dim3 blocks(16, 16); 
        dim3 grid((m_screenWidth + 15) / 16, (m_screenHeight + 15) / 16);

        renderKernel<<<grid, blocks>>>(surf, m_screenWidth, m_screenHeight);
        err = hipGetLastError();
        if (err != hipSuccess) {
            std::cerr << "CUDA Kernel Error 6: " << hipGetErrorString(err) << "\n";
        }

        hipDeviceSynchronize();
        err = hipGetLastError();
        if (err != hipSuccess) {
            std::cerr << "CUDA Kernel Error 7: " << hipGetErrorString(err) << "\n";
        }

        hipDestroySurfaceObject(surf);
        hipGraphicsUnmapResources(1, &m_texPtr, 0);

    }
}